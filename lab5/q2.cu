#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__device__ int getGTID() {
    int blockid = blockIdx.x + blockIdx.y * gridDim.x + blockIdx.z * gridDim.x * gridDim.y;
    int threadid = blockid * (blockDim.x * blockDim.y * blockDim.z) + 
                   (threadIdx.z * blockDim.x * blockDim.y) + 
                   (threadIdx.y * blockDim.x) + 
                   threadIdx.x;
    return threadid;
}


__global__ void add(int *a,int *b, int *c,int n){
    int gtid=getGTID();
    if(gtid< n)
        c[gtid]=a[gtid]+b[gtid];

}


int main(){

    int *a,*b,*c;
	int *d_A, *d_B, *d_C;
   	printf("Enter no. of elements: ");
	int  N;
	scanf("%d",&N);
    	int S = N * sizeof(int);
	a = (int*)malloc(S);
	b = (int*)malloc(S);
	c = (int*)malloc(S);
	hipMalloc((void**)&d_A , S);
	hipMalloc((void**)&d_B , S);
	hipMalloc((void**)&d_C , S);

printf("220905440\n");
    printf("Enter elements in A: ");
	for(int i=0; i<N; i++)
	{
		a[i]=rand()%10;
        printf("%d \t",a[i]);
	}
    printf("\n");

    printf("Enter elements in B: ");
	for(int i=0; i<N; i++)
	{
		b[i]=rand()%10;
        printf("%d \t",b[i]);
	}

	hipMemcpy(d_A,a,S,hipMemcpyHostToDevice);
	hipMemcpy(d_B,b,S,hipMemcpyHostToDevice);
       int blocks = (int)ceil((float)N / 256); 
	add<<<blocks,256>>>(d_A,d_B,d_C,N);

	hipMemcpy(c,d_C,S,hipMemcpyDeviceToHost);
   printf("\n");
   printf("No. of blocks is %d\n",blocks);
      printf("No. of unused threads %d\n",blocks*256-N);
	printf("Result: ");
	for(int i=0; i<N; i++)
	{
		printf("%d ",c[i]);
	}
	printf("\n");

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	free(a);
	free(b);
	free(c);
	return 0;
    



}